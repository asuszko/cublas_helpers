#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "cu_errchk.h"
#include "cublas_copy.h"


template<typename T>
inline hipblasStatus_t cublasTcopy(hipblasHandle_t *handle,
                                  int n,
                                  const T *x, int incx,
                                  T *y, int incy)
{
    if (std::is_same<T, float>::value) {
        return hipblasScopy(*handle, n,
                          (float *)x, incx,
                          (float *)y, incy);
    }
    else
    if (std::is_same<T, double>::value) {
        return hipblasDcopy(*handle, n,
                          (double *)x, incx,
                          (double *)y, incy);
    }
    else
    if (std::is_same<T, hipComplex>::value) {
        return hipblasCcopy(*handle, n,
                          (hipComplex *)x, incx,
                          (hipComplex *)y, incy);
    }
    else
    if (std::is_same<T, hipDoubleComplex>::value) {
        return hipblasZcopy(*handle, n,
                          (hipDoubleComplex *)x, incx,
                          (hipDoubleComplex *)y, incy);
    }
    else {
        return HIPBLAS_STATUS_EXECUTION_FAILED;
    }
}


/**
*  http://docs.nvidia.com/cuda/cublas/index.html#cublas-lt-t-gt-copy
*/
void cublas_copy(hipblasHandle_t *handle,
                 int n,
                 const void *x, int incx,
                 void *y, int incy,
                 int dtype)
{

    switch(dtype) {
        case 0: {
            gpuBlasErrchk(cublasTcopy(handle,
                                      n,
                                      (float*)x, incx,
                                      (float*)y, incy));
            break;
        }
        case 1: {
            gpuBlasErrchk(cublasTcopy(handle,
                                      n,
                                      (double*)x, incx,
                                      (double*)y, incy));
            break;
        }
        case 2: {
            gpuBlasErrchk(cublasTcopy(handle,
                                      n,
                                      (hipComplex*)x, incx,
                                      (hipComplex*)y, incy));
             break;
        }
        case 3: {
            gpuBlasErrchk(cublasTcopy(handle,
                                      n,
                                      (hipDoubleComplex*)x, incx,
                                      (hipDoubleComplex*)y, incy));
            break;
        }
    }

    return;
}
