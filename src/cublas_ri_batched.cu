#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "cu_errchk.h"
#include "cublas_ri_batched.h"



void cublas_ri_batched(hipblasHandle_t *handle,
                       int n,
                       void *d_A[],
                       int *PivotArray,
                       void *d_C[],
                       int *infoArray,
                       int batchSize,
                       int dtype)
{
    switch(dtype) {

        case 0:
        {
            // float *ASptr = static_cast<float*>(*Aarray);
            // gpuBlasErrchk(hipblasSgetrfBatched(*handle, n, 
                                              // &ASptr, n, 
                                              // PivotArray,
                                              // infoArray,
                                              // batchSize));
            // break;
        }

        case 1:
        {
            // double *ADptr = static_cast<double*>(*Aarray);
            // gpuBlasErrchk(hipblasDgetrfBatched(*handle, n, 
                                              // &ADptr, n, 
                                              // PivotArray,
                                              // infoArray,
                                              // batchSize));
            // break;
        }
        
        case 2:
        {
            //float2 *ACptr = static_cast<float2*>(*Aarray);
            gpuBlasErrchk(hipblasCgetriBatched(*handle, n,
                                              (const float2 **)d_A, n,
                                              PivotArray,
                                              (float2 **)d_C, n,
                                              infoArray,
                                              batchSize));
            break;
        }
        
        case 3:
        {
            // double2 *AZptr = static_cast<double2*>(*Aarray);
            // gpuBlasErrchk(hipblasZgetrfBatched(*handle, n, 
                                              // &AZptr, n, 
                                              // PivotArray,
                                              // infoArray,
                                              // batchSize));
            // break;
        }
    }

    return;

}
