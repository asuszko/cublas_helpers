#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "cu_errchk.h"
#include "cublas_gemm.h"


/**
*  http://docs.nvidia.com/cuda/cublas/index.html#cublas-lt-t-gt-gemm
*/
void cublas_gemm(hipblasHandle_t *handle,
                 hipblasOperation_t transa,
                 hipblasOperation_t transb,
                 int m, int n, int k,
                 void *alpha,
                 void *d_A, int lda,
                 void *d_B, int ldb,
                 void *beta,
                 void *d_C, int ldc,
                 int dtype)
{

    switch(dtype) {

        case 0:
            gpuBlasErrchk(hipblasSgemm(*handle,
                                      transa,transb,
                                      m,n,k,
                                      static_cast<const float*>(alpha),
                                      static_cast<const float*>(d_A), lda,
                                      static_cast<const float*>(d_B), ldb,
                                      static_cast<const float*>(beta),
                                      static_cast<float*>(d_C), ldc));
            break;

        case 1:
            gpuBlasErrchk(hipblasDgemm(*handle,
                                      transa,transb,
                                      m,n,k,
                                      static_cast<const double*>(alpha),
                                      static_cast<const double*>(d_A), lda,
                                      static_cast<const double*>(d_B), ldb,
                                      static_cast<const double*>(beta),
                                      static_cast<double*>(d_C), ldc));
            break;

        case 2:
            gpuBlasErrchk(hipblasCgemm(*handle,
                                      transa,transb,
                                      m,n,k,
                                      static_cast<const float2*>(alpha),
                                      static_cast<const float2*>(d_A), lda,
                                      static_cast<const float2*>(d_B), ldb,
                                      static_cast<const float2*>(beta),
                                      static_cast<float2*>(d_C), ldc));
            break;

        case 3:
            gpuBlasErrchk(hipblasZgemm(*handle,
                                      transa,transb,
                                      m,n,k,
                                      static_cast<const double2*>(alpha),
                                      static_cast<const double2*>(d_A), lda,
                                      static_cast<const double2*>(d_B), ldb,
                                      static_cast<const double2*>(beta),
                                      static_cast<double2*>(d_C), ldc));
            break;
    }

    return;
}
