#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "cu_errchk.h"
#include "cublas_gemm.h"


template<typename T>
inline hipblasStatus_t cublasTgemm(hipblasHandle_t *handle,
      hipblasOperation_t transa, hipblasOperation_t transb,
      int m, int n, int k,
      const T *alpha,
      const T *A, int lda,
      const T *B, int ldb,
      const T *beta,
      T *C, int ldc)
{
    if (std::is_same<T, float>::value) {
        return hipblasSgemm(*handle, transa, transb, m, n, k,
                          (float *)alpha, (float *)A, lda,
                          (float *)B, ldb, (float *)beta,
                          (float *)C, ldc);
    }
    else
    if (std::is_same<T, double>::value) {
        return hipblasDgemm(*handle, transa, transb, m, n, k,
                          (double *)alpha, (double *)A, lda,
                          (double *)B, ldb, (double *)beta,
                          (double *)C, ldc);
    }
    else
    if (std::is_same<T, hipComplex>::value) {
        return hipblasCgemm(*handle, transa, transb, m, n, k,
                          (hipComplex *)alpha, (hipComplex *)A, lda,
                          (hipComplex *)B, ldb, (hipComplex *)beta,
                          (hipComplex *)C, ldc);
    }
    else
    if (std::is_same<T, hipDoubleComplex>::value) {
        return hipblasZgemm(*handle, transa, transb, m, n, k,
                          (hipDoubleComplex *)alpha, (hipDoubleComplex *)A, lda,
                          (hipDoubleComplex *)B, ldb, (hipDoubleComplex *)beta,
                          (hipDoubleComplex *)C, ldc);
    }
    else {
        return HIPBLAS_STATUS_EXECUTION_FAILED;
    }
}




/* C compatible version that requires a dtype_id to be converted
to the proper data type. */
void cublas_gemm(hipblasHandle_t *handle,
                 hipblasOperation_t transa,
                 hipblasOperation_t transb,
                 int m, int n, int k,
                 const void *alpha,
                 const void *d_A, int lda,
                 const void *d_B, int ldb,
                 const void *beta,
                 void *d_C, int ldc,
                 int dtype)
{

    switch(dtype) {

        case 0: {
            gpuBlasErrchk(cublasTgemm(handle,
                                      transa,transb,
                                      m,n,k,
                                      (float*)alpha,
                                      (float*)d_A, lda,
                                      (float*)d_B, ldb,
                                      (float*)beta,
                                      (float*)d_C, ldc));
            break;
        }

        case 1: {
            gpuBlasErrchk(cublasTgemm(handle,
                                      transa,transb,
                                      m,n,k,
                                      (double*)alpha,
                                      (double*)d_A, lda,
                                      (double*)d_B, ldb,
                                      (double*)beta,
                                      (double*)d_C, ldc));
            break;
        }

        case 2: {
            gpuBlasErrchk(cublasTgemm(handle,
                                      transa,transb,
                                      m,n,k,
                                      (hipComplex*)alpha,
                                      (hipComplex*)d_A, lda,
                                      (hipComplex*)d_B, ldb,
                                      (hipComplex*)beta,
                                      (hipComplex*)d_C, ldc));
            break;
        }

        case 3: {
            gpuBlasErrchk(cublasTgemm(handle,
                                      transa,transb,
                                      m,n,k,
                                      (hipDoubleComplex*)alpha,
                                      (hipDoubleComplex*)d_A, lda,
                                      (hipDoubleComplex*)d_B, ldb,
                                      (hipDoubleComplex*)beta,
                                      (hipDoubleComplex*)d_C, ldc));
            break;
        }
    }

    return;
}
