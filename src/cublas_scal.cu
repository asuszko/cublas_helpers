#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "cu_errchk.h"
#include "cublas_scal.h"


template<typename T>
inline hipblasStatus_t cublasTscal(hipblasHandle_t *handle,
        int n,
        const T *alpha,
        T *x, int incx)
{
    if (std::is_same<T, float>::value) {
        return hipblasSscal(*handle, n,
                          (float *)alpha,
                          (float *)x, incx);
    }
    else
    if (std::is_same<T, double>::value) {
        return hipblasDscal(*handle, n,
                          (double *)alpha,
                          (double *)x, incx);
    }
    else
    if (std::is_same<T, hipComplex>::value) {
        return hipblasCscal(*handle, n,
                          (hipComplex *)alpha,
                          (hipComplex *)x, incx);
    }
    else
    if (std::is_same<T, hipDoubleComplex>::value) {
        return hipblasZscal(*handle, n,
                          (hipDoubleComplex *)alpha,
                          (hipDoubleComplex *)x, incx);
    }
    else {
        return HIPBLAS_STATUS_EXECUTION_FAILED;
    }
}



/**
*  http://docs.nvidia.com/cuda/cublas/index.html#cublas-lt-t-gt-scal
*/
void cublas_scal(hipblasHandle_t *handle,
                 int n,
                 void *alpha,
                 void *d_x, int incx,
                 int dtype)
{
    switch(dtype) {

        case 0: {
            gpuBlasErrchk(cublasTscal(handle, n,
                                      (float*)alpha,
                                      (float*)d_x, incx));
            break;
        }

        case 1: {
            gpuBlasErrchk(cublasTscal(handle, n,
                                      (double*)alpha,
                                      (double*)d_x, incx));
            break;
        }

        case 2: {
            gpuBlasErrchk(cublasTscal(handle, n,
                                      (hipComplex*)alpha,
                                      (hipComplex*)d_x, incx));
            break;
        }

        case 3: {
            gpuBlasErrchk(cublasTscal(handle, n,
                                      (hipDoubleComplex*)alpha,
                                      (hipDoubleComplex*)d_x, incx));
            break;
        }
    }

    return;

}
