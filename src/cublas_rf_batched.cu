#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "cu_errchk.h"
#include "cublas_rf_batched.h"



void cublas_rf_batched(hipblasHandle_t *handle,
                       int n,
                       void *Aarray[],
                       int *PivotArray,
                       int *infoArray,
                       int batchSize,
                       int dtype)
{
    switch(dtype) {

        case 0:
        {
            gpuBlasErrchk(hipblasSgetrfBatched(*handle, n, 
                                              reinterpret_cast<float**>(Aarray), n, 
                                              PivotArray,
                                              infoArray,
                                              batchSize));
            break;
        }

        case 1:
        {
            gpuBlasErrchk(hipblasDgetrfBatched(*handle, n, 
                                              reinterpret_cast<double**>(Aarray), n, 
                                              PivotArray,
                                              infoArray,
                                              batchSize));
            break;
        }
        
        case 2:
        {
            gpuBlasErrchk(hipblasCgetrfBatched(*handle, n, 
                                              reinterpret_cast<float2**>(Aarray), n, 
                                              PivotArray,
                                              infoArray,
                                              batchSize));
            break;
        }
        
        case 3:
        {
            gpuBlasErrchk(hipblasZgetrfBatched(*handle, n, 
                                              reinterpret_cast<double2**>(Aarray), n, 
                                              PivotArray,
                                              infoArray,
                                              batchSize));
            break;
        }
    }

    return;

}
