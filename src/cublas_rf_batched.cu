#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "cu_errchk.h"
#include "cublas_rf_batched.h"



void cublas_rf_batched(hipblasHandle_t *handle,
                       int n,
                       void *Aarray[],
                       int *PivotArray,
                       int *infoArray,
                       int batchSize,
                       int dtype)
{
    switch(dtype) {

        case 0:
        {
            // float *ASptr = static_cast<float*>(*Aarray);
            gpuBlasErrchk(hipblasSgetrfBatched(*handle, n, 
                                              (float**)Aarray, n, 
                                              PivotArray,
                                              infoArray,
                                              batchSize));
            break;
        }

        case 1:
        {
            // double *ADptr = static_cast<double*>(*Aarray);
            gpuBlasErrchk(hipblasDgetrfBatched(*handle, n, 
                                              (double**)Aarray, n, 
                                              PivotArray,
                                              infoArray,
                                              batchSize));
            break;
        }
        
        case 2:
        {
            //float2 *ACptr = static_cast<float2*>(*Aarray);
            gpuBlasErrchk(hipblasCgetrfBatched(*handle, n, 
                                              (float2**)Aarray, n, 
                                              PivotArray,
                                              infoArray,
                                              batchSize));
            break;
        }
        
        case 3:
        {
            // double2 *AZptr = static_cast<double2*>(*Aarray);
            gpuBlasErrchk(hipblasZgetrfBatched(*handle, n, 
                                              (double2**)Aarray, n, 
                                              PivotArray,
                                              infoArray,
                                              batchSize));
            break;
        }
    }

    return;

}
