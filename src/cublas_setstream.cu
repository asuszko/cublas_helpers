#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include "cu_errchk.h"
#include "cublas_setstream.h"

void cublas_setstream(hipblasHandle_t *handle, hipStream_t *stream)
{
    if(stream == NULL) {
        gpuBlasErrchk(hipblasSetStream(*handle,NULL));
    }
    else {
        gpuBlasErrchk(hipblasSetStream(*handle,*stream));
    }
    return;
}
