#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include "cu_errchk.h"
#include "cublas_setstream.h"

/**
*  Set the stream for the cuBLAS handle.
*  @param handle - [hipblasHandle_t*] : The cuBLAS handle.
*  @param stream - [hipStream_t*] : CUDA stream.
*/
void cublas_setstream(hipblasHandle_t *handle, hipStream_t *stream)
{
    if(stream == NULL) {
        gpuBlasErrchk(hipblasSetStream(*handle,NULL));
    }
    else {
        gpuBlasErrchk(hipblasSetStream(*handle,*stream));
    }
    return;
}
