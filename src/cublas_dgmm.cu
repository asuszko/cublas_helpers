#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "cu_errchk.h"
#include "cublas_dgmm.h"


template<typename T>
inline hipblasStatus_t cublasTdgmm(hipblasHandle_t *handle,
                                  hipblasSideMode_t mode,
                                  int m, int n,
                                  const T *A, int lda,
                                  const T *x, int incx,
                                  T *C, int ldc)
{
    if (std::is_same<T, float>::value) {
        return hipblasSdgmm(*handle, mode, m, n,
                          (float *)A, lda,
                          (float *)x, incx,
                          (float *)C, ldc);
    }
    else
    if (std::is_same<T, double>::value) {
        return hipblasDdgmm(*handle, mode, m, n,
                          (double *)A, lda,
                          (double *)x, incx,
                          (double *)C, ldc);
    }
    else
    if (std::is_same<T, hipComplex>::value) {
        return hipblasCdgmm(*handle, mode, m, n,
                          (hipComplex *)A, lda,
                          (hipComplex *)x, incx,
                          (hipComplex *)C, ldc);
    }
    else
    if (std::is_same<T, hipDoubleComplex>::value) {
        return hipblasZdgmm(*handle, mode, m, n,
                          (hipDoubleComplex *)A, lda,
                          (hipDoubleComplex *)x, incx,
                          (hipDoubleComplex *)C, ldc);
    }
    else {
        return HIPBLAS_STATUS_EXECUTION_FAILED;
    }
}




/* C compatible version that requires a dtype_id to be converted
to the proper data type. */
void cublas_dgmm(hipblasHandle_t *handle,
                 hipblasSideMode_t mode,
                 int m, int n,
                 const void *d_A, int lda,
                 const void *d_x, int incx,
                 void *d_C, int ldc,
                 int dtype)
{

    switch(dtype) {

        case 0: {
            gpuBlasErrchk(cublasTdgmm(handle,
                                      mode,
                                      m, n,
                                      (float*)d_A, lda,
                                      (float*)d_x, incx,
                                      (float*)d_C, ldc));
            break;
        }

        case 1: {
            gpuBlasErrchk(cublasTdgmm(handle,
                                      mode,
                                      m, n,
                                      (double*)d_A, lda,
                                      (double*)d_x, incx,
                                      (double*)d_C, ldc));
            break;
        }

        case 2: {
            gpuBlasErrchk(cublasTdgmm(handle,
                                      mode,
                                      m, n,
                                      (hipComplex*)d_A, lda,
                                      (hipComplex*)d_x, incx,
                                      (hipComplex*)d_C, ldc));
            break;
        }

        case 3: {
            gpuBlasErrchk(cublasTdgmm(handle,
                                      mode,
                                      m, n,
                                      (hipDoubleComplex*)d_A, lda,
                                      (hipDoubleComplex*)d_x, incx,
                                      (hipDoubleComplex*)d_C, ldc));
            break;
        }
    }

    return;
}
