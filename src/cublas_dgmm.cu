#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "cu_errchk.h"
#include "cublas_dgmm.h"


/**
*  http://docs.nvidia.com/cuda/cublas/index.html#cublas-lt-t-gt-dgmm
*/
void cublas_dgmm(hipblasHandle_t *handle,
                 hipblasSideMode_t mode,
                 int m, int n,
                 void *d_A, int lda,
                 void *d_x, int incx,
                 void *d_C, int ldc,
                 int dtype)
{

    switch(dtype) {

        case 0:
            gpuBlasErrchk(hipblasSdgmm(*handle, mode, m, n,
                                      static_cast<float*>(d_A), lda,
                                      static_cast<float*>(d_x), incx,
                                      static_cast<float*>(d_C), ldc));
            break;

        case 1:
            gpuBlasErrchk(hipblasDdgmm(*handle, mode, m, n,
                                      static_cast<double*>(d_A), lda,
                                      static_cast<double*>(d_x), incx,
                                      static_cast<double*>(d_C), ldc));
            break;

        case 2:
            gpuBlasErrchk(hipblasCdgmm(*handle, mode, m, n,
                                      static_cast<float2*>(d_A), lda,
                                      static_cast<float2*>(d_x), incx,
                                      static_cast<float2*>(d_C), ldc));
            break;

        case 3:
            gpuBlasErrchk(hipblasZdgmm(*handle, mode, m, n,
                                      static_cast<double2*>(d_A), lda,
                                      static_cast<double2*>(d_x), incx,
                                      static_cast<double2*>(d_C), ldc));
            break;

    }

    return;
}
