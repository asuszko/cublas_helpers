
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "cu_errchk.h"
#include "cublas_init.h"


/**
 *  Initialize a cuBLAS library context.
 *  @return handle - [hipblasHandle_t*] - cuBLAS handle
 */
hipblasHandle_t *cublas_init()
{
    /* Create cuBLAS handle. */
    hipblasHandle_t *handle = (hipblasHandle_t*)malloc(sizeof(hipblasHandle_t));

    /* Initialize cuBLAS library context. */
    gpuBlasErrchk(hipblasCreate(handle));

    /* Return pointer to the handle. */
    return handle;
}


void cublas_destroy(hipblasHandle_t *handle)
{
    gpuBlasErrchk(hipblasDestroy(*handle))
    return;
}
