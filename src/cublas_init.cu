#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "cu_errchk.h"
#include "cublas_init.h"

/**
*  Initialize cuBLAS handle.
*  @return [hipblasHandle_t *] : cuBLAS handle.
*/
hipblasHandle_t *cublas_init()
{
    /* Create cuBLAS handle. */
    hipblasHandle_t *handle = new hipblasHandle_t;

    /* Initialize cuBLAS library context. */
    gpuBlasErrchk(hipblasCreate(handle));

    /* Return pointer to the handle. */
    return handle;
}

/**
*  Destroy cuBLAS handle.
*  @param [hipblasHandle_t *] : cuBLAS handle.
*/
void cublas_destroy(hipblasHandle_t *handle)
{
    gpuBlasErrchk(hipblasDestroy(*handle))
    delete[] handle;
    return;
}
