#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "cu_errchk.h"
#include "cublas_nrm2.h"


template<typename T>
inline hipblasStatus_t cublasTnrm2(hipblasHandle_t *handle,
                                  int n,
                                  const T *x, int incx,
                                  void *result)
{
    if (std::is_same<T, float>::value) {
        return hipblasSnrm2(*handle, n,
                          (float *)x, incx,
                          (float *)result);
    }
    else
    if (std::is_same<T, double>::value) {
        return hipblasDnrm2(*handle, n,
                          (double *)x, incx,
                          (double *)result);
    }
    else
    if (std::is_same<T, hipComplex>::value) {
        return hipblasScnrm2(*handle, n,
                           (hipComplex *)x, incx,
                           (float *)result);
    }
    else
    if (std::is_same<T, hipDoubleComplex>::value) {
        return hipblasDznrm2(*handle, n,
                           (hipDoubleComplex *)x, incx,
                           (double *)result);
    }
    else {
        return HIPBLAS_STATUS_EXECUTION_FAILED;
    }
}



/**
*  http://docs.nvidia.com/cuda/cublas/index.html#cublas-lt-t-gt-nrm2
*/
void cublas_nrm2(hipblasHandle_t *handle,
                 int n,
                 const void *x, int incx,
                 void *result,
                 int dtype)
{

    switch(dtype) {
        case 0: {
            gpuBlasErrchk(cublasTnrm2(handle,
                                      n,
                                      (float *)x, incx,
                                      (float *)result));
            break;
        }
        case 1: {
            gpuBlasErrchk(cublasTnrm2(handle,
                                      n,
                                      (double *)x, incx,
                                      (double *)result));
            break;
        }
        case 2: {
            gpuBlasErrchk(cublasTnrm2(handle,
                                      n,
                                      (hipComplex *)x, incx,
                                      (float *)result));
            break;
        }
        case 3: {
            gpuBlasErrchk(cublasTnrm2(handle,
                                      n,
                                      (hipDoubleComplex *)x, incx,
                                      (double *)result));
            break;
        }
    }

    return;
}
