#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "cu_errchk.h"
#include "cublas_axpy.h"


template<typename T>
inline hipblasStatus_t cublasTaxpy(hipblasHandle_t *handle,
        int n,
        const T *alpha,
        const T *x, int incx,
        T *y, int incy)
{
    if (std::is_same<T, float>::value) {
        return hipblasSaxpy(*handle, n, (float *)alpha,
                          (float *)x, incx,
                          (float *)y, incy);
    }
    else
    if (std::is_same<T, double>::value) {
        return hipblasDaxpy(*handle, n, (double *)alpha,
                          (double *)x, incx,
                          (double *)y, incy);
    }
    else
    if (std::is_same<T, hipComplex>::value) {
        return hipblasCaxpy(*handle, n, (hipComplex *)alpha,
                          (hipComplex *)x, incx,
                          (hipComplex *)y, incy);
    }
    else
    if (std::is_same<T, hipDoubleComplex>::value) {
        return hipblasZaxpy(*handle, n, (hipDoubleComplex *)alpha,
                          (hipDoubleComplex *)x, incx,
                          (hipDoubleComplex *)y, incy);
    }
    else {
        return HIPBLAS_STATUS_EXECUTION_FAILED;
    }
}



/* C compatible version that requires a dtype_id to be converted
to the proper data type. */
void cublas_axpy(hipblasHandle_t *handle,
                 int n,
                 const void *alpha,
                 const void *x, int incx,
                 void *y, int incy,
                 int dtype)
{

    switch(dtype) {
        case 0: {
            gpuBlasErrchk(cublasTaxpy(handle,
                                      n,
                                      (float *)alpha,
                                      (float *)x, incx,
                                      (float *)y, incy));
            break;
        }
        case 1: {
            gpuBlasErrchk(cublasTaxpy(handle,
                                      n,
                                      (double *)alpha,
                                      (double *)x, incx,
                                      (double *)y, incy));
            break;
        }
        case 2: {
            gpuBlasErrchk(cublasTaxpy(handle,
                                      n,
                                      (hipComplex *)alpha,
                                      (hipComplex *)x, incx,
                                      (hipComplex *)y, incy));
            break;
        }
        case 3: {
            gpuBlasErrchk(cublasTaxpy(handle,
                                      n,
                                      (hipDoubleComplex *)alpha,
                                      (hipDoubleComplex *)x, incx,
                                      (hipDoubleComplex *)y, incy));
            break;
        }
    }

    return;
}
