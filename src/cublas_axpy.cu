#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "cu_errchk.h"
#include "cublas_axpy.h"


/**
*  http://docs.nvidia.com/cuda/cublas/index.html#cublas-lt-t-gt-axpy
*/
void cublas_axpy(hipblasHandle_t *handle,
                 int n,
                 void *alpha,
                 void *x, int incx,
                 void *y, int incy,
                 int dtype)
{

    switch(dtype) {
        case 0:
            gpuBlasErrchk(hipblasSaxpy(*handle, n,
                                      static_cast<float*>(alpha),
                                      static_cast<float*>(x), incx,
                                      static_cast<float*>(y), incy));
            break;

        case 1:
            gpuBlasErrchk(hipblasDaxpy(*handle, n,
                                      static_cast<double*>(alpha),
                                      static_cast<double*>(x), incx,
                                      static_cast<double*>(y), incy));
            break;

        case 2:
            gpuBlasErrchk(hipblasCaxpy(*handle, n,
                                      static_cast<float2*>(alpha),
                                      static_cast<float2*>(x), incx,
                                      static_cast<float2*>(y), incy));
            break;

        case 3:
            gpuBlasErrchk(hipblasZaxpy(*handle, n,
                                      static_cast<double2*>(alpha),
                                      static_cast<double2*>(x), incx,
                                      static_cast<double2*>(y), incy));
            break;

    }

    return;
}
